
#include <hip/hip_runtime.h>
#include <stdio.h>

template<int Order>
__global__ void test_kernel() {
    printf("Running kernel of order %d\n", Order);
}

template<int Order>
void launch() {
    test_kernel<Order><<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
}

template void launch<2>();


int main(){
    launch<1>();
    return 0;
}
